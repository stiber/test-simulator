/*
 * DynamicSpikingSynapse_struct_d.cu
 * CUDA side struct of DynamicSpikingSynapse
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


/**
 * Allocate data members in the allocSynapseStruct_d.
 * @param count
 */
void allocSynapseStruct_d( int count ) {
	DynamicSpikingSynapse_struct synapse;

	if ( count > 0 ) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.inUse, count * sizeof( bool ) ) );
		HANDLE_ERROR( hipMemset( synapse.inUse, 0, count * sizeof( bool ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.summationPoint, count * sizeof( PFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.summationCoord, count * sizeof( Coordinate ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.synapseCoord, count * sizeof( Coordinate ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.deltaT, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.W, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.psr, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.decay, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.total_delay, count * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.type, count * sizeof( synapseType ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.delayQueue, count * sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.ldelayQueue, count * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.tau, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.r, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.u, count * sizeof( FLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &synapse.lastSpike, count * sizeof( uint64_t ) ) );

		HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( synapse_st_d), &synapse, sizeof( DynamicSpikingSynapse_struct ) ) );
	}
}

/**
 * Deallocate data members in the DynamicSpikingSynapse_struct_d
 */
void deleteSynapseStruct_d( ) {
	DynamicSpikingSynapse_struct synapse;

	HANDLE_ERROR( hipMemcpyFromSymbol( &synapse, HIP_SYMBOL(synapse_st_d), sizeof( DynamicSpikingSynapse_struct ) ) );

	HANDLE_ERROR( hipFree( synapse.inUse ) );
	HANDLE_ERROR( hipFree( synapse.summationPoint ) );
	HANDLE_ERROR( hipFree( synapse.summationCoord ) );
	HANDLE_ERROR( hipFree( synapse.synapseCoord ) );
	HANDLE_ERROR( hipFree( synapse.deltaT ) );
	HANDLE_ERROR( hipFree( synapse.W ) );
	HANDLE_ERROR( hipFree( synapse.psr ) );
	HANDLE_ERROR( hipFree( synapse.decay ) );
	HANDLE_ERROR( hipFree( synapse.total_delay ) );
	HANDLE_ERROR( hipFree( synapse.type ) );
	HANDLE_ERROR( hipFree( synapse.delayQueue ) );
	HANDLE_ERROR( hipFree( synapse.ldelayQueue ) );
	HANDLE_ERROR( hipFree( synapse.tau ) );
	HANDLE_ERROR( hipFree( synapse.r ) );
	HANDLE_ERROR( hipFree( synapse.u ) );
	HANDLE_ERROR( hipFree( synapse.lastSpike ) );
}

/**
 * Copy DynamicSpikingSynapse_struct data for GPU processing.
 * @param synapse_h
 * @param count
 */
void copySynapseHostToDevice( DynamicSpikingSynapse_struct& synapse_h, int count ) {
	// copy everything necessary
	DynamicSpikingSynapse_struct synapse;

	if ( count > 0 ) {
        	HANDLE_ERROR( hipMemcpyFromSymbol( &synapse, HIP_SYMBOL(synapse_st_d), sizeof( DynamicSpikingSynapse_struct ) ) );

		HANDLE_ERROR( hipMemcpy ( synapse.inUse, synapse_h.inUse, count * sizeof( bool ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.summationCoord, synapse_h.summationCoord, count * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.synapseCoord, synapse_h.synapseCoord, count * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.summationPoint, synapse_h.summationPoint, count * sizeof( PFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.deltaT, synapse_h.deltaT, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.W, synapse_h.W, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.psr, synapse_h.psr, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.decay, synapse_h.decay, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.total_delay, synapse_h.total_delay, count * sizeof( int ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.type, synapse_h.type, count * sizeof( synapseType ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.delayQueue, synapse_h.delayQueue, count * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.ldelayQueue, synapse_h.ldelayQueue, count * sizeof( int ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.r, synapse_h.r, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.u, synapse_h.u, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.tau, synapse_h.tau, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( synapse.lastSpike, synapse_h.lastSpike, count * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
	}
}

/**
 * Copy data from GPU into DynamicSpikingSynapse_struct.
 * @param synapse_h
 * @param count
 */
void copySynapseDeviceToHost( DynamicSpikingSynapse_struct& synapse_h, int count ) {
	// copy everything necessary
	DynamicSpikingSynapse_struct synapse;

	if ( count > 0 ) {
        	HANDLE_ERROR( hipMemcpyFromSymbol( &synapse, HIP_SYMBOL(synapse_st_d), sizeof( DynamicSpikingSynapse_struct ) ) );

		HANDLE_ERROR( hipMemcpy ( synapse_h.inUse, synapse.inUse, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.summationCoord, synapse.summationCoord, count * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.synapseCoord, synapse.synapseCoord, count * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.deltaT, synapse.deltaT, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.W, synapse.W, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.psr, synapse.psr, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.decay, synapse.decay, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.total_delay, synapse.total_delay, count * sizeof( int ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.type, synapse.type, count * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.r, synapse.r, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.u, synapse.u, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.tau, synapse.tau, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.delayQueue, synapse.delayQueue, count * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.lastSpike, synapse.lastSpike, count * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
	}
}

void copySynapseSumCoordDeviceToHost( DynamicSpikingSynapse_struct& synapse_h, int count ) {
	// copy everything necessary
	DynamicSpikingSynapse_struct synapse;

	if ( count > 0 ) {
        	HANDLE_ERROR( hipMemcpyFromSymbol( &synapse, HIP_SYMBOL(synapse_st_d), sizeof( DynamicSpikingSynapse_struct ) ) );

		HANDLE_ERROR( hipMemcpy ( synapse_h.inUse, synapse.inUse, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.summationCoord, synapse.summationCoord, count * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
	}
}
