/*
 * LifNeuron_struct_d.cu
 * CUDA side struct of LifNeuron
 */

/**
 * Allocate data members in the LifNeuron_struct_d.
 * @param count
 */
void allocNeuronStruct_d( int count ) {
	LifNeuron_struct neuron;

	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.deltaT, count * sizeof( double ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.summationPoint, count * sizeof( PFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Cm, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Rm, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Vthresh, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Vrest, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Vreset, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Vinit, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Trefract, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Inoise, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.randNoise, count * sizeof( PFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Iinject, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Isyn, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.nStepsInRefr, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.C1, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.C2, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.I0, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Vm, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.hasFired, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.Tau, count * sizeof( FLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.spikeCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.outgoingSynapse_begin, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.synapseCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.incomingSynapse_begin, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.inverseCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.numNeurons, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &neuron.stepDuration, count * sizeof( int ) ) );
	
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( neuron_st_d), &neuron, sizeof( LifNeuron_struct ) ) );
}

/**
 * Deallocate data members in the LifNeuron_struct_d.
 */
void deleteNeuronStruct_d(  ) {
	LifNeuron_struct neuron;
	HANDLE_ERROR( hipMemcpyFromSymbol( &neuron, HIP_SYMBOL(neuron_st_d), sizeof( LifNeuron_struct ) ) );

	HANDLE_ERROR( hipFree( neuron.deltaT ) );
	HANDLE_ERROR( hipFree( neuron.summationPoint ) );
	HANDLE_ERROR( hipFree( neuron.Cm ) );
	HANDLE_ERROR( hipFree( neuron.Rm ) );
	HANDLE_ERROR( hipFree( neuron.Vthresh ) );
	HANDLE_ERROR( hipFree( neuron.Vrest ) );
	HANDLE_ERROR( hipFree( neuron.Vreset ) );
	HANDLE_ERROR( hipFree( neuron.Vinit ) );
	HANDLE_ERROR( hipFree( neuron.Trefract ) );
	HANDLE_ERROR( hipFree( neuron.Inoise ) );
	HANDLE_ERROR( hipFree( neuron.randNoise ) );
	HANDLE_ERROR( hipFree( neuron.Iinject ) );
	HANDLE_ERROR( hipFree( neuron.Isyn ) );
	HANDLE_ERROR( hipFree( neuron.nStepsInRefr ) );
	HANDLE_ERROR( hipFree( neuron.C1 ) );
	HANDLE_ERROR( hipFree( neuron.C2 ) );
	HANDLE_ERROR( hipFree( neuron.I0 ) );
	HANDLE_ERROR( hipFree( neuron.Vm ) );
	HANDLE_ERROR( hipFree( neuron.hasFired ) );
	HANDLE_ERROR( hipFree( neuron.Tau ) );
	HANDLE_ERROR( hipFree( neuron.spikeCount ) );
	HANDLE_ERROR( hipFree( neuron.outgoingSynapse_begin ) );
	HANDLE_ERROR( hipFree( neuron.synapseCount ) );
	HANDLE_ERROR( hipFree( neuron.incomingSynapse_begin ) );
	HANDLE_ERROR( hipFree( neuron.inverseCount ) );
	HANDLE_ERROR( hipFree( neuron.numNeurons ) );
	HANDLE_ERROR( hipFree( neuron.stepDuration ) );
}

/**
 * Copy LifNeuron_struct data for GPU processing.
 * @param neuron_h
 * @param count
 */
void copyNeuronHostToDevice( LifNeuron_struct& neuron_h, int count ) {
	LifNeuron_struct neuron;
	HANDLE_ERROR( hipMemcpyFromSymbol( &neuron, HIP_SYMBOL(neuron_st_d), sizeof( LifNeuron_struct ) ) );

	HANDLE_ERROR( hipMemcpy ( neuron.deltaT, neuron_h.deltaT, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.summationPoint, neuron_h.summationPoint, count * sizeof( PFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Cm, neuron_h.Cm, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Rm, neuron_h.Rm, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Vthresh, neuron_h.Vthresh, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Vrest, neuron_h.Vrest, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Vreset, neuron_h.Vreset, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Vinit, neuron_h.Vinit, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Trefract, neuron_h.Trefract, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Inoise, neuron_h.Inoise, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Iinject, neuron_h.Iinject, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Isyn, neuron_h.Isyn, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.nStepsInRefr, neuron_h.nStepsInRefr, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.C1, neuron_h.C1, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.C2, neuron_h.C2, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.I0, neuron_h.I0, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Vm, neuron_h.Vm, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.hasFired, neuron_h.hasFired, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.Tau, neuron_h.Tau, count * sizeof( FLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.spikeCount, neuron_h.spikeCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.outgoingSynapse_begin, neuron_h.outgoingSynapse_begin, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.synapseCount, neuron_h.synapseCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.incomingSynapse_begin, neuron_h.incomingSynapse_begin, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( neuron.inverseCount, neuron_h.inverseCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
}

/**
 * Copy data from GPU into LifNeuron_struct.
 * @param neuron_h
 * @param count
 */
void copyNeuronDeviceToHost( LifNeuron_struct& neuron_h, int count ) {
	LifNeuron_struct neuron;
	HANDLE_ERROR( hipMemcpyFromSymbol( &neuron, HIP_SYMBOL(neuron_st_d), sizeof( LifNeuron_struct ) ) );

	HANDLE_ERROR( hipMemcpy ( neuron_h.C1, neuron.C1, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.C2, neuron.C2, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Cm, neuron.C1, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.I0, neuron.I0, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Iinject, neuron.Iinject, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Inoise, neuron.Inoise, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Isyn, neuron.Isyn, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Rm, neuron.Rm, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Tau, neuron.Tau, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Trefract, neuron.Trefract, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Vinit, neuron.Vinit, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Vm, neuron.Vm, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Vrest, neuron.Vrest, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Vreset, neuron.Vreset, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.Vthresh, neuron.Vthresh, count * sizeof( FLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.nStepsInRefr, neuron.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.spikeCount, neuron.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_h.synapseCount, neuron.synapseCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
}
